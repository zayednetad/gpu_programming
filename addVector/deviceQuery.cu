#include <iostream>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h" // Include hip/hip_runtime_api.h for _ConvertSMVer2Cores

int main() {
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?" << std::endl;
        return 1;
    }

    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return 1;
    }

    std::cout << "Number of CUDA devices: " << deviceCount << std::endl;

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        std::cout << "\nDevice " << i << " properties:" << std::endl;
        std::cout << "  Device name: " << deviceProp.name << std::endl;
        std::cout << "  Compute capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total global memory: " << deviceProp.totalGlobalMem << " bytes" << std::endl;
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  CUDA cores per multiprocessor: " << (deviceProp.multiProcessorCount * _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor)) << std::endl;
        std::cout << "  Clock rate: " << deviceProp.clockRate << " kHz" << std::endl;
    }

    return 0;
}
