// This program computer the sum of two N-element vectors using unified memory
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cassert>
#include <iostream>

using std::cout;

// CUDA kernel for vector addition
// No change when using CUDA unified memory
__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    // Calculate global thread thread ID
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    // Each thread processes a portion of the array
    int start = tid * (N / blockDim.x);
    int end = start + (N / blockDim.x);

    // Ensure that we don't go beyond the array size
    end = min(end, N);

    for (int i = start; i < end; ++i) {
        c[i] = a[i] + b[i];
    }
}

int main() {
  // Array size of 2^16 (65536 elements)
  //const int N = 1 << 16;
  const int N = 1000000000; // Number of elements in the array
  size_t bytes = N * sizeof(int);

  // Declare unified memory pointers
  int *a, *b, *c;

  // Allocation memory for these pointers
  hipMallocManaged(&a, bytes);
  hipMallocManaged(&b, bytes);
  hipMallocManaged(&c, bytes);

  // Initialize vectors
  for (int i = 0; i < N; i++) {
    a[i] = rand() % 100;
    b[i] = rand() % 100;
  }

  // Threads per CTA (1024 threads per CTA)
  int BLOCK_SIZE = 1 << 10;

  // CTAs per Grid
  int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

  // Call CUDA kernel
  vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c, N);

  // Wait for all previous operations before using values
  // We need this because we don't get the implicit synchronization of
  // cudaMemcpy like in the original example
  hipDeviceSynchronize();

  // Verify the result on the CPU
  //for (int i = 0; i < N; i++) {
  //  assert(c[i] == a[i] + b[i]);
  //}

  // Free unified memory (same as memory allocated with cudaMalloc)
  hipFree(a);
  hipFree(b);
  hipFree(c);

  cout << "COMPLETED SUCCESSFULLY!\n";

  return 0;
}
